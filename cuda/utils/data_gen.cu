#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <data_gen.h>
#include <stdio.h>


__global__ void simple_1d_gen(float *a, float *b, int n)
{
    //a bit of an overkill to do this on the device
    // but the name of directory is cuda
    //  :)
    // For debugging, printing, and other purposes better to do it on host rather
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        a[idx] = idx;
        b[idx] = idx  + 1.0f;
    }
}